#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include "scan_common.h"

//All three kernels run 512 threads per workgroup
//Must be a power of two
#define THREADBLOCK_SIZE 512
#define LOOP_PERTHREAD 16
#define LOOP_PERTHREAD2 16

////////////////////////////////////////////////////////////////////////////////
// Basic ccan codelets
////////////////////////////////////////////////////////////////////////////////
//Naive inclusive scan: O(N * log2(N)) operations
//Allocate 2 * 'size' local memory, initialize the first half
//with 'size' zeros avoiding if(pos >= offset) condition evaluation
//and saving instructions

inline __device__ uint scan1Inclusive(uint idata, volatile uint *s_Data, uint size)
{
    uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
    s_Data[pos] = 0;
    pos += size;
    s_Data[pos] = idata;

    for (uint offset = 1; offset < size; offset <<= 1)
    {
        __syncthreads();
        uint t = s_Data[pos] + s_Data[pos - offset];
        __syncthreads();
        s_Data[pos] = t;
    }

    return s_Data[pos];
}

inline __device__ uint scan1Exclusive(uint idata, volatile uint *s_Data, uint size)
{
    return scan1Inclusive(idata, s_Data, size) - idata;
}


inline __device__ uint4 scan4Inclusive(uint4 idata4, volatile uint *s_Data, uint size)
{
    //Level-0 inclusive scan
    idata4.y += idata4.x;
    idata4.z += idata4.y;
    idata4.w += idata4.z;

    //Level-1 exclusive scan
    uint oval = scan1Exclusive(idata4.w, s_Data, size / 4);

    idata4.x += oval;
    idata4.y += oval;
    idata4.z += oval;
    idata4.w += oval;

    return idata4;
}

//Exclusive vector scan: the array to be scanned is stored
//in local thread memory scope as uint4
inline __device__ uint4 scan4Exclusive(uint4 idata4, volatile uint *s_Data, uint size)
{
    uint4 odata4 = scan4Inclusive(idata4, s_Data, size);
    odata4.x -= idata4.x;
    odata4.y -= idata4.y;
    odata4.z -= idata4.z;
    odata4.w -= idata4.w;
    return odata4;
}

////////////////////////////////////////////////////////////////////////////////
// Scan kernels
////////////////////////////////////////////////////////////////////////////////
__global__ void scanExclusiveShared(
    uint4 *d_Dst,
    uint4 *d_Src,
    uint size
)
{
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load data
    uint4 idata4 = d_Src[pos];

    //Calculate exclusive scan
    uint4 odata4 = scan4Exclusive(idata4, s_Data, size);

    //Write back
    d_Dst[pos] = odata4;
}

//Exclusive scan of top elements of bottom-level scans (4 * THREADBLOCK_SIZE)
__global__ void scanExclusiveShared2(
    uint *d_Buf,
    uint *d_Dst,
    uint *d_Src,
    uint N,
    uint arrayLength
)
{
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    //Skip loads and stores for inactive threads of last threadblock (pos >= N)
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load top elements
    //Convert results of bottom-level scan back to inclusive
    uint idata = 0;

    if (pos < N)
        idata =
            d_Dst[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos] + d_Src[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos];

    //Compute
    uint odata = scan1Exclusive(idata, s_Data, arrayLength);

    //Avoid out-of-bound access
    if (pos < N)
    {
        d_Buf[pos] = odata;
    }
}

__global__ void scanExclusiveShared3(
                                     uint *e_Buf,
                                     uint *d_Buf,
                                     uint *d_Dst,
                                     uint *d_Src,
                                     uint N,
                                     uint arrayLength
                                     )
{
  __shared__ uint s_Data[2 * THREADBLOCK_SIZE];
  
  //Skip loads and stores for inactive threads of last threadblock (pos >= N)
  uint pos = blockIdx.x * blockDim.x + threadIdx.x;
  
  //Load top elements
  //Convert results of bottom-level scan back to inclusive
  uint idata = 0;
  
  if (pos < N)
    idata =
      d_Buf[THREADBLOCK_SIZE -1 + pos * THREADBLOCK_SIZE] + d_Dst[(4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE) * pos] + d_Src[(4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE) * pos];
  
  //Compute
  uint odata = scan1Exclusive(idata, s_Data, arrayLength);
  
  //Avoid out-of-bound access
  if (pos < N)
    {
      e_Buf[pos] = odata;
    }
}


//Final step of large-array scan: combine basic inclusive scan with exclusive scan of top elements of input arrays
__global__ void uniformUpdate(
    uint4 *d_Data,
    uint *d_Buffer
)
{
    __shared__ uint buf;
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIdx.x == 0)
    {
        buf = d_Buffer[blockIdx.x];
    }

    __syncthreads();

    uint4 data4 = d_Data[pos];
    data4.x += buf;
    data4.y += buf;
    data4.z += buf;
    data4.w += buf;
    d_Data[pos] = data4;
}

__global__ void uniformUpdate2(
    uint4 *d_Data,
    uint *d_Buffer
)
{
    __shared__ uint buf;
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    uint temp = blockIdx.x/THREADBLOCK_SIZE;
    if (threadIdx.x == 0)
    {
        buf = d_Buffer[temp];
    }

    __syncthreads();

    uint4 data4 = d_Data[pos];
    data4.x += buf;
    data4.y += buf;
    data4.z += buf;
    data4.w += buf;
    d_Data[pos] = data4;
}

__global__ void diff_kernel(
    uint *d_Data,
    uint *d_Src,
    uint pnum,
    uint length,
    uint size
)
{

  uint pos = blockIdx.x * blockDim.x + threadIdx.x;
  uint p_n = pnum;
  uint len = length;
  uint POS = pos * LOOP_PERTHREAD;
  uint i;

  for(i = POS ; (i < POS + LOOP_PERTHREAD)&&(i < len-1); i++){      
    d_Data[i] = d_Src[(i+1)*p_n] - d_Src[i * p_n];          
  }

  if(i == (len-1)){
    d_Data[len-1] = size - d_Src[(len-1)*p_n];
  }

}


__global__ void transport_kernel(
    uint *d_Data,
    uint *d_Src,
    uint loc

)
{

  d_Data[0] = d_Src[loc-1];


}



////////////////////////////////////////////////////////////////////////////////
// Interface function
////////////////////////////////////////////////////////////////////////////////
//Derived as 32768 (max power-of-two gridDim.x) * 4 * THREADBLOCK_SIZE
//Due to scanExclusiveShared<<<>>>() 1D block addressing
extern "C" const uint MAX_BATCH_ELEMENTS = 4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE * THREADBLOCK_SIZE;
extern "C" const uint MIN_SHORT_ARRAY_SIZE = 4;
extern "C" const uint MAX_SHORT_ARRAY_SIZE = 4 * THREADBLOCK_SIZE;
extern "C" const uint MIN_LARGE_ARRAY_SIZE = 8 * THREADBLOCK_SIZE;
extern "C" const uint MAX_LARGE_ARRAY_SIZE = 4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE;
extern "C" const uint MIN_LL_SIZE = 8 * THREADBLOCK_SIZE * THREADBLOCK_SIZE;
extern "C" const uint MAX_LL_SIZE = MAX_BATCH_ELEMENTS;//4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE * THREADBLOCK_SIZE;


//Internal exclusive scan buffer
static uint *d_Buf;
static uint *e_Buf;

extern "C" void initScan(void)
{

  hipMalloc((void **)&d_Buf, (MAX_BATCH_ELEMENTS / (4 * THREADBLOCK_SIZE)) * sizeof(uint));
  
  checkCudaErrors(hipMalloc((void **)&e_Buf, (MAX_BATCH_ELEMENTS / (4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE)) * sizeof(uint)));

}

extern "C" void closeScan(void)
{
    checkCudaErrors(hipFree(d_Buf));
    checkCudaErrors(hipFree(e_Buf));
    
}

static uint factorRadix2(uint &log2L, uint L)
{
    if (!L)
    {
        log2L = 0;
        return 0;
    }
    else
    {
        for (log2L = 0; (L & 1) == 0; L >>= 1, log2L++);

        return L;
    }
}

static uint iDivUp(uint dividend, uint divisor)
{
    return ((dividend % divisor) == 0) ? (dividend / divisor) : (dividend / divisor + 1);
}

extern "C" size_t scanExclusiveShort(
    uint *d_Dst,
    uint *d_Src,
    uint arrayLength
)
{
    //Check power-of-two factorization
    uint log2L;
    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
    assert(factorizationRemainder == 1);

    //Check supported size range
    assert((arrayLength >= MIN_SHORT_ARRAY_SIZE) && (arrayLength <= MAX_SHORT_ARRAY_SIZE));

    //Check total batch size limit
    assert(arrayLength <= MAX_BATCH_ELEMENTS);

    //Check all threadblocks to be fully packed with data
    assert(arrayLength % (4 * THREADBLOCK_SIZE) == 0);

    scanExclusiveShared<<<arrayLength / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint4 *)d_Src,
        arrayLength
    );
    getLastCudaError("scanExclusiveShared() execution FAILED\n");

    return THREADBLOCK_SIZE;
}

extern "C" size_t scanExclusiveLarge(
    uint *d_Dst,
    uint *d_Src,
    uint arrayLength
)
{
    //Check power-of-two factorization
  /*
    uint log2L;
    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
    assert(factorizationRemainder == 1);
  */
    assert(arrayLength%MAX_SHORT_ARRAY_SIZE == 0);

    //Check supported size range
    assert((arrayLength >= MIN_LARGE_ARRAY_SIZE) && (arrayLength <= MAX_LARGE_ARRAY_SIZE));

    //Check total batch size limit
    assert(arrayLength <= MAX_BATCH_ELEMENTS);

    scanExclusiveShared<<<arrayLength / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint4 *)d_Src,
        4 * THREADBLOCK_SIZE
    );
    getLastCudaError("scanExclusiveShared() execution FAILED\n");

    //Not all threadblocks need to be packed with input data:
    //inactive threads of highest threadblock just don't do global reads and writes

    uint array_temp = THREADBLOCK_SIZE;
    for(uint i = 2; i<=THREADBLOCK_SIZE ; i <<= 1){
      if(i >= arrayLength/(4 * THREADBLOCK_SIZE)){
        array_temp = i;
        break;
      }
    }

    const uint blockCount2 = 1;//iDivUp((batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE);
    scanExclusiveShared2<<< blockCount2, THREADBLOCK_SIZE>>>(
        (uint *)d_Buf,
        (uint *)d_Dst,
        (uint *)d_Src,
        arrayLength / (4 * THREADBLOCK_SIZE),
        array_temp
    );
    getLastCudaError("scanExclusiveShared2() execution FAILED\n");

    uniformUpdate<<<(arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint *)d_Buf
    );
    getLastCudaError("uniformUpdate() execution FAILED\n");

    return THREADBLOCK_SIZE;
}

extern "C" size_t scanExclusiveLL(
    uint *d_Dst,
    uint *d_Src,
    uint arrayLength
)
{
    //Check power-of-two factorization
  /*
    uint log2L;
    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
    assert(factorizationRemainder == 1);
  */
  assert((arrayLength%MAX_LARGE_ARRAY_SIZE) == 0);

    //Check supported size range
    assert((arrayLength >= MIN_LL_SIZE) && (arrayLength <= MAX_LL_SIZE));

    //Check total batch size limit
    assert((arrayLength) <= MAX_BATCH_ELEMENTS);

    scanExclusiveShared<<<arrayLength / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint4 *)d_Src,
        4 * THREADBLOCK_SIZE
    );
    getLastCudaError("scanExclusiveShared() execution FAILED\n");
    checkCudaErrors(hipDeviceSynchronize());

    //Now ,prefix sum per THREADBLOCK_SIZE done


    //Not all threadblocks need to be packed with input data:
    //inactive threads of highest threadblock just don't do global reads and writes

    const uint blockCount2 = iDivUp (arrayLength / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE);
    scanExclusiveShared2<<< blockCount2, THREADBLOCK_SIZE>>>(
        (uint *)d_Buf,
        (uint *)d_Dst,
        (uint *)d_Src,
        arrayLength / (4 * THREADBLOCK_SIZE),
        THREADBLOCK_SIZE
    );
    getLastCudaError("scanExclusiveShared2() execution FAILED\n");
    checkCudaErrors(hipDeviceSynchronize());


    //prefix sum of last elements per THREADBLOCK_SIZE done
    //this prefix sum can caluculate under only THREADBLOCK_SIZE size.
    //so We need one more prefix sum for last elements.

    uint array_temp = THREADBLOCK_SIZE;
    for(uint i = 2; i<=THREADBLOCK_SIZE ; i <<= 1){
      if(i >= arrayLength/(4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE)){
        array_temp = i;
        break;
      }
    }

    const uint blockCount3 = 1;//(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE);
    scanExclusiveShared3<<< blockCount3, THREADBLOCK_SIZE>>>(
        (uint *)e_Buf,
        (uint *)d_Buf,
        (uint *)d_Dst,
        (uint *)d_Src,
        arrayLength / (4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE),
        array_temp
    );
    getLastCudaError("scanExclusiveShared3() execution FAILED\n");
    checkCudaErrors(hipDeviceSynchronize());


    //add d_Buf to each array of d_Dst
    uniformUpdate<<<arrayLength / (4 * THREADBLOCK_SIZE ), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint *)d_Buf
    );

    //add e_Buf to each array of d_Dst
    checkCudaErrors(hipDeviceSynchronize());

    uniformUpdate2<<<arrayLength / (4 * THREADBLOCK_SIZE ), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint *)e_Buf
    );
    getLastCudaError("uniformUpdate() execution FAILED\n");

    checkCudaErrors(hipDeviceSynchronize());
    return THREADBLOCK_SIZE;
}


extern "C" size_t diff_Part(
    uint *d_Dst,
    uint *d_Src,
    uint diff,
    uint arrayLength,
    uint size
)
{

    const uint blockCount = iDivUp(arrayLength , LOOP_PERTHREAD*THREADBLOCK_SIZE);
    diff_kernel<<<blockCount, THREADBLOCK_SIZE>>>(
        d_Dst,
        d_Src,
        diff,
        arrayLength,
        size
    );
    getLastCudaError("diff_Part() execution FAILED\n");
    checkCudaErrors(hipDeviceSynchronize());

    return THREADBLOCK_SIZE;
}


//transport input data to output per diff
extern "C" void transport_gpu(
    uint *d_Dst,
    uint *d_Src,
    uint loc
)
{

    //Check total batch size limit
    //assert((arrayLength) <= MAX_BATCH_ELEMENTS);

  const uint blockCount = 1;//iDivUp(arrayLength , LOOP_PERTHREAD2*THREADBLOCK_SIZE);
  transport_kernel<<<blockCount, 1>>>(
                                                     d_Dst,
                                                     d_Src,
                                                     loc
                                                     );
  getLastCudaError("transport_gpu() execution FAILED\n");
  checkCudaErrors(hipDeviceSynchronize());

}
