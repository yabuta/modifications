#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "GPUTUPLE.h"
#include "GPUetc/common/GNValue.h"
#include "GPUetc/expressions/Gcomparisonexpression.h"

using namespace voltdb;

extern "C" {

  /**
     called function is changed by join condition.
     
     if T1.val = T2.val, iocount and iojoin is called.
     if T.val1 = T.val2 , iicount and iijoin is called.
   */


__global__
void count(
          COLUMNDATA *oCD,
          COLUMNDATA *iCD,
          GComparisonExpression ex,
          int *count,
          int ltn,
          int rtn
          ) 

{

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int k = blockIdx.y * gridDim.x * blockDim.x;

  if(i<ltn){

    __shared__ COLUMNDATA tiCD[BLOCK_SIZE_Y];
    if(threadIdx.x==0){
      for(uint j=0 ; j<BLOCK_SIZE_Y && BLOCK_SIZE_Y*blockIdx.y+j<rtn ; j++){
        tiCD[j] = iCD[BLOCK_SIZE_Y*blockIdx.y + j];
      }
    }

    __syncthreads();

    COLUMNDATA toCD=oCD[i];
    int rtn_g = rtn;
    int mcount = 0;
    for(uint j = 0; j<BLOCK_SIZE_Y && BLOCK_SIZE_Y*blockIdx.y+j<rtn_g;j++){
      if(ex.eval(toCD.gn,tiCD[j].gn)) {
        mcount++;
      }     
    }

    count[i+k] = mcount;
  }

  if(i+k == (blockDim.x*gridDim.x*gridDim.y-1)){
    count[i+k+1] = 0;
  }

}


__global__ void join(
          COLUMNDATA *oCD,
          COLUMNDATA *iCD,
          RESULT *p,
          GComparisonExpression ex,
          int *count,
          int ltn,
          int rtn,
          int ll,
          int rr
          ) 
{

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int k = blockIdx.y * gridDim.x * blockDim.x;

  if(i<ltn){
    __shared__ COLUMNDATA tiCD[BLOCK_SIZE_Y];
    if(threadIdx.x==0){
      for(uint j=0 ; j<BLOCK_SIZE_Y && BLOCK_SIZE_Y*blockIdx.y+j<rtn ; j++){
        tiCD[j] = iCD[BLOCK_SIZE_Y*blockIdx.y + j];
      }
    }
    __syncthreads();

    COLUMNDATA toCD = oCD[i];
    int rtn_g = rtn;
    int writeloc = count[i+k];
    for(uint j = 0; j<BLOCK_SIZE_Y && BLOCK_SIZE_Y*blockIdx.y+j<rtn_g;j++){
      if(ex.eval(toCD.gn,tiCD[j].gn)){
        p[writeloc].lkey = toCD.num;
        p[writeloc].rkey = tiCD[j].num;
        writeloc++;
      }
    }
  }     
}

}
